
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace::std;

#define root_2 1.414213562373095048801688724209

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int *data_img;

__device__ int *query_img;

__device__ float *candidate_img;

class pixel {

	public:

	int i,j;
	int r,g,b;
	
	__device__  pixel(int row, int col) {
	
		i = row;
		j = col;
		
	}
	
	__device__ void setRGB(int *img, int m, int n) {
	
		r = img[i*n*3 + j*3 + 0];
		g = img[i*n*3 + j*3 + 1];
		b = img[i*n*3 + j*3 + 2];
		
	}
	
};

struct matchVal {

	float rmsd;
	int x,y;
	int angle;
	
	matchVal(float rmsd_val, int x_val, int y_val, int angle_val) {
	
		rmsd = rmsd_val;
		x = x_val;
		y = y_val;
		angle = angle_val;
	}

};

struct CompareVal {
    bool operator()(matchVal const& p1, matchVal const& p2)
    {
        return p1.rmsd < p2.rmsd;
    }
};


__device__  float RMSD(int m, int n, int i0, int j0, int search_m, int search_n, int *dataImg, int *queryImg) {
		
	//here dataImg and queryImg are different from global data_img and query_img respectively
	// m,n - dimensions of dataImg
	// search_m, search_n - dimensions of queryImg == (search window)
	// lc_x, lc_y - left corner row, col in dataImg
	
	float total = 0;

	for(int i=0; i<search_m; i++) 
	{
		for(int j=0; j<search_n; j++) 
		{
			for(int k=0; k<3; k++) 
			{
				float diff = dataImg[(i+i0)*n*3 + (j+j0)*3 + k] - queryImg[i*search_n*3 + j*3 + k];
				total += diff*diff;
			}
		
		}
	
	}
	
	return sqrt(total/(search_m*search_n*3));
}

__device__ float calcGrayValue_dev(int m, int n, int lx, int ly, int box_m, int box_n) 
{
	// no need to pass img as argument as img is always data_img
	// lx, ly - left corner row and col
	// m, n - dimensions of the image
	// box_m, box_n - dimensions of search box

	float total = 0;
	int count = 0;

	for(int i=lx; i< lx+box_m; i++) 
	{
		for(int j=ly; j< ly+box_n; j++) 
		{
	 		for(int k=0; k<3; k++) 
	 		{
	 		 	total += data_img[i*n*3 + j*3 + k];
	 		 	count++;
	 		}	
		}
	}
	return total/count;
}

__host__ float calcGrayValue_host(int *img, int m, int n, int lx, int ly, int box_m, int box_n) 
{

	// lx, ly - left corner row and col
	// m, n - dimensions of the image
	// box_m, box_n - dimensions of search box

    float total = 0;
    int count = 0;

    for(int i=lx; i< lx+box_m; i++) 
    {
    	for(int j=ly; j< ly+box_n; j++) 
    	{
       		for(int k=0; k<3; k++) 
       		{
       		 	total += img[i*n*3 + j*3 + k];
       		 	count++;
       		}	
    	}
    }

    return total/count;
}

__global__ void kernel0(int i_start, int i_end, int j_start, int j_end, int m1, int n1, int m2, int n2, float th1, float th2, float gray_val_of_query)
{
	
   int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
   int i = tid/n1;
   int j = tid - n1*(tid/n1);
   candidate_img[2*i*n1+2*j] = -1;
	candidate_img[2*i*n1+2*j+1] = -1;	

   if (i >= i_start && i < i_end && j >= j_start && j < j_end)
   {
		float gray_val_of_box = calcGrayValue_dev(m1, n1, i, j, m2, n2);

		if (abs(gray_val_of_query - gray_val_of_box) <= th2)
		{
			float rmsd = RMSD(m1, n1, i, j, m2, n2, data_img, query_img);
			if(rmsd <= th1) 
			{
				// printf("Vertical, Found at lc = (%d,%d) with dist = %f\n", i, j, rmsd);
				candidate_img[2*i*n1+2*j] = rmsd;
				candidate_img[2*i*n1+2*j+1] = 0;
			}
		}
	} 
}

__global__ void kernel1(int i_start, int i_end, int j_start, int j_end, int m1, int n1, int m2, int n2, float th1, float th2, float gray_val_of_query)
{

	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
   int i = tid/n1;
   int j = tid - n1*(tid/n1);
   candidate_img[2*i*n1+2*j] = -1;
	candidate_img[2*i*n1+2*j+1] = -1;	

   //i and j are the indexes of the lowest point
   if (i >= i_start && i < i_end && j >= j_start && j < j_end)
   {
		int lx = i;
		int ly = j-floor(m2/root_2);
		int box_m = floor((m2+n2)/root_2)+1;
		int box_n = floor(n2/root_2) + floor(m2/root_2)+1;

		float gray_val_of_box = calcGrayValue_dev(m1, n1, lx, ly, box_m, box_n);
		
		if (abs(gray_val_of_query - gray_val_of_box) <= th2)
		{
			float rmsd = 0;
			// printf("i: %d, j: %d, th1: %f, th2: %f, diff: %f\n",i, j, th1, th2, abs(gray_val_of_query - gray_val_of_box) );

			for (int i1 = 0; i1 < m2; i1++)
			{
				for (int j1 = 0; j1 < n2; j1++)
				{
					float x = i + i1/root_2 + j1/root_2;
					float y = j - i1/root_2 + j1/root_2;

					pixel A(floor(x), floor(y));
					pixel B(A.i, A.j+1);
					pixel C(A.i+1, A.j+1);
					pixel D(A.i+1, A.j);
					
					A.setRGB(data_img,m1,n1);
					B.setRGB(data_img,m1,n1);
					C.setRGB(data_img,m1,n1);
					D.setRGB(data_img,m1,n1);
					
					pixel res(-1,-1);
					
					float delta_y = x - floor(x);
					float delta_x = y - floor(y);
					
					res.r = A.r*(1-delta_x)*(1-delta_y) + B.r*delta_x*(1-delta_y) + D.r*(1-delta_x)*delta_y + C.r*delta_x*delta_y;
					res.g = A.g*(1-delta_x)*(1-delta_y) + B.g*delta_x*(1-delta_y) + D.g*(1-delta_x)*delta_y + C.g*delta_x*delta_y;
					res.b = A.b*(1-delta_x)*(1-delta_y) + B.b*delta_x*(1-delta_y) + D.b*(1-delta_x)*delta_y + C.b*delta_x*delta_y;
					
					float delta_r = (query_img[i1*n2*3+j1*3+0]-res.r);
					float delta_g = (query_img[i1*n2*3+j1*3+1]-res.g);
					float delta_b = (query_img[i1*n2*3+j1*3+2]-res.b);

					rmsd += delta_r*delta_r + delta_g*delta_g + delta_b*delta_b;
				}
			}

			rmsd = sqrt(rmsd/(m2*n2*3));
			if (rmsd <= th1) 
			{
				// printf("Angle 45, Found at lc = (%d,%d) with dist = %f\n", i, j, rmsd);	
				candidate_img[2*i*n1+2*j] = rmsd;
				candidate_img[2*i*n1+2*j+1] = 45;
			}
		}
	}
}

__global__ void kernel2(int i_start, int i_end, int j_start, int j_end, int m1, int n1, int m2, int n2, float th1, float th2, float gray_val_of_query)
{
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
   int i = tid/n1;
   int j = tid - n1*(tid/n1);
   candidate_img[2*i*n1+2*j] = -1;
	candidate_img[2*i*n1+2*j+1] = -1;	

   //i and j are the indexes of the lowest point
   if (i >= i_start && i < i_end && j >= j_start && j < j_end)
   {
		int lx = i - floor(n2/root_2);
		int ly = j;
		int box_m = floor(n2/root_2) + floor(m2/root_2)+1;
		int box_n = floor((m2+n2)/root_2)+1;
		float gray_val_of_box = calcGrayValue_dev(m1, n1, lx, ly, box_m, box_n);
		
		if (abs(gray_val_of_query - gray_val_of_box) <= th2)
		{
			float rmsd = 0;
			// printf("i: %d, j: %d, th1: %f, th2: %f, diff: %f\n",i, j, th1, th2, abs(gray_val_of_query - gray_val_of_box) );

			for (int i1 = 0; i1 < m2; i1++)
			{
				for (int j1 = 0; j1 < n2; j1++)
				{
					float x = i + i1/root_2 - j1/root_2;
					float y = j + i1/root_2 + j1/root_2;

					pixel A(floor(x), floor(y));
					pixel B(A.i, A.j+1);
					pixel C(A.i+1, A.j+1);
					pixel D(A.i+1, A.j);
					
					A.setRGB(data_img,m1,n1);
					B.setRGB(data_img,m1,n1);
					C.setRGB(data_img,m1,n1);
					D.setRGB(data_img,m1,n1);
					
					pixel res(-1,-1);
					
					float delta_y = x - floor(x);
					float delta_x = y - floor(y);
					
					res.r = A.r*(1-delta_x)*(1-delta_y) + B.r*delta_x*(1-delta_y) + D.r*(1-delta_x)*delta_y + C.r*delta_x*delta_y;
					res.g = A.g*(1-delta_x)*(1-delta_y) + B.g*delta_x*(1-delta_y) + D.g*(1-delta_x)*delta_y + C.g*delta_x*delta_y;
					res.b = A.b*(1-delta_x)*(1-delta_y) + B.b*delta_x*(1-delta_y) + D.b*(1-delta_x)*delta_y + C.b*delta_x*delta_y;
					
					float delta_r = (query_img[i1*n2*3+j1*3+0]-res.r);
					float delta_g = (query_img[i1*n2*3+j1*3+1]-res.g);
					float delta_b = (query_img[i1*n2*3+j1*3+2]-res.b);

					rmsd += delta_r*delta_r + delta_g*delta_g + delta_b*delta_b;
				}
			}

			rmsd = sqrt(rmsd/(m2*n2*3));
			if (rmsd <= th1) 
			{
				// printf("Angle -45, Found at lc = (%d,%d) with dist = %f\n", i, j, rmsd);
				candidate_img[2*i*n1+2*j] = rmsd;
				candidate_img[2*i*n1+2*j+1] = -45;		
			}
		}
	}
}

//case when image is +45 rotated
__host__ void searchRotated1(int *dataImg, int m1, int n1, int *queryImg, int m2, int n2, float th1, float th2)
{

	float gray_val_of_query = calcGrayValue_host(queryImg, m2, n2, 0, 0, m2, n2);

	int i_start = 0;
	int i_end = m1 - floor((m2+n2)/root_2) - 1;

	int j_start = ceil(m2/root_2);
	int j_end = n1 - floor(n2/root_2) - 1;

	dim3 dimBlock(32, 32);
	dim3 dimGrid((m1*n1)/1024 + 1, 1);

	kernel1<<<dimGrid , dimBlock>>>(i_start, i_end, j_start, j_end, m1, n1, m2, n2, th1, th2, gray_val_of_query);
	hipDeviceSynchronize();
}

//case when image is -45 rotated
__host__ void searchRotated2(int *dataImg, int m1, int n1, int *queryImg, int m2, int n2, float th1, float th2)
{
	float gray_val_of_query = calcGrayValue_host(queryImg, m2, n2, 0, 0, m2, n2);

	int i_start = ceil(n2/root_2);
	int i_end = m1 - floor(m2/root_2) - 1;

	int j_start = 0;
	int j_end = n1 - floor((m2+n2)/root_2) - 1;

	dim3 dimBlock(32, 32);
	dim3 dimGrid((m1*n1)/1024 + 1, 1);

	kernel2<<<dimGrid , dimBlock>>>(i_start, i_end, j_start, j_end, m1, n1, m2, n2, th1, th2, gray_val_of_query);
	hipDeviceSynchronize();
}


__host__ void searchVertical(int *dataImg, int m1, int n1, int *queryImg, int m2, int n2, float th1, float th2) {
	
	//here i and j are lower left corners
	float gray_val_of_query = calcGrayValue_host(queryImg, m2, n2, 0, 0, m2, n2);

	int i_start = 0, i_end = m1-m2+1;
	int j_start = 0, j_end = n1-n2+1;

	dim3 dimBlock(32, 32);
	dim3 dimGrid((m1*n1)/1024 + 1, 1);

	kernel0<<<dimGrid , dimBlock>>>(i_start, i_end, j_start, j_end, m1, n1, m2, n2, th1, th2, gray_val_of_query);
	hipDeviceSynchronize();

}

__host__ int* readImage(string &filename, int &m, int &n) {

	ifstream file;
	file.open(filename.c_str());

	int x;
	file >> m;
	file >> n;

	int *img = (int*)malloc(3*m*n*sizeof(int));	
	for (int i = 0; i < m; i++)
    {
    	for (int j = 0; j < n; j++)
    	{
    		for (int k = 0; k < 3; k++)
    		{
    			file >> x;
    			img[(m-i-1)*n*3 + j*3 + k] = x;
    		}
    	}
    }

	file.close();
	return img;
}

void getTopMatches(priority_queue<matchVal, vector<matchVal>, CompareVal> &pq, float *candidates, int m1, int n1, int n, bool finalcheck) {

	for(int i=0; i<2*m1*n1; i+=2) 
	{
		
		matchVal currVal(candidates[i], i/(2*n1), (i%(2*n1))/2, (int)candidates[i+1]);

		if(currVal.rmsd < 0) continue;
			
		if(pq.size() < n) 
		{
			pq.push(currVal);	
		}
		else 
		{
			//here topVal has maximum rmsd
			matchVal topVal = pq.top();
				
			if(topVal.rmsd < currVal.rmsd) 
			{
				continue;
			}
			pq.pop();
			pq.push(currVal);
		}
	}

	vector<matchVal> result;
	
	while(!pq.empty() && finalcheck) {
		
		matchVal curr = pq.top();
		//cout << curr.rmsd << " " << curr.x << " " << curr.y << " " << curr.angle << "\n";
		result.push_back(curr);
		pq.pop();
	}
	if (finalcheck)
	{
		ofstream myfile;
		myfile.open("output.txt");
		for (int i = result.size()-1; i >= 0; i--)
		{
			myfile << result[i].x;
			myfile << " ";
			myfile << result[i].y;
			myfile << " ";
			myfile << result[i].angle;
			myfile << "\n";
		}
		myfile.close();
	}
}

int main(int argc, char** argv) 
{
	// th1 => threshold for RMSD
	// th2 => threshold for GrayValue 

	string imagefile = string(argv[1]);
	string queryfile = string(argv[2]);
	float th1=stof(argv[3]), th2=stof(argv[4]);
	int n = stoi(argv[5]);

	int m1,n1;
	int *dataImg = readImage(imagefile, m1, n1);

	int m2, n2;
	int *queryImg = readImage(queryfile, m2, n2);


	int *d_data;
	gpuErrchk(hipMalloc(&d_data, 3*m1*n1*sizeof(int)));
	gpuErrchk(hipMemcpy(d_data, &dataImg[0], 3*m1*n1*sizeof(int),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(data_img), &d_data, sizeof(int*)));

   
	int *d_query;
	gpuErrchk(hipMalloc(&d_query, 3*m2*n2*sizeof(int)));
	gpuErrchk(hipMemcpy(d_query, &queryImg[0], 3*m2*n2*sizeof(int),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(query_img), &d_query, sizeof(int*)));

	float *candidates;
	candidates = (float*)malloc(2*m1*n1*sizeof(float));
	for (int i = 0; i < 2*m1*n1; i++) candidates[i] = -1;

	float *d_candidates;
	gpuErrchk(hipMalloc(&d_candidates, 2*m1*n1*sizeof(float)));
	gpuErrchk(hipMemcpy(d_candidates, &candidates[0], 2*m1*n1*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(candidate_img), &d_candidates, sizeof(float*)));

	priority_queue<matchVal, vector<matchVal>, CompareVal> pq;

	searchVertical(dataImg, m1, n1, queryImg, m2, n2, th1, th2);
	gpuErrchk(hipMemcpy(candidates, &d_candidates[0], 2*m1*n1*sizeof(float), hipMemcpyDeviceToHost));
	getTopMatches(pq, candidates, m1, n1, n, 0);

	searchRotated1(dataImg, m1, n1, queryImg, m2, n2, th1, th2);
	gpuErrchk(hipMemcpy(candidates, &d_candidates[0], 2*m1*n1*sizeof(float), hipMemcpyDeviceToHost));
	getTopMatches(pq, candidates, m1, n1, n, 0);

	searchRotated2(dataImg, m1, n1, queryImg, m2, n2, th1, th2);
	gpuErrchk(hipMemcpy(candidates, &d_candidates[0], 2*m1*n1*sizeof(float), hipMemcpyDeviceToHost));
	getTopMatches(pq, candidates, m1, n1, n, 1);   

	
	hipFree(d_query);
	hipFree(d_candidates);
	hipFree(d_data);

	return 0;
}
